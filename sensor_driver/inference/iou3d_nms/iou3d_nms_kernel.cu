/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

// #define DEBUG
const int THREADS_PER_BLOCK_NMS = sizeof(unsigned long long) * 8;
const float EPS = 1e-8;
struct Point {
    float x, y;
    __device__ Point() {}
    __device__ Point(double _x, double _y){
        x = _x, y = _y;
    }

    __device__ void set(float _x, float _y){
        x = _x; y = _y;
    }

    __device__ Point operator +(const Point &b)const{
        return Point(x + b.x, y + b.y);
    }

    __device__ Point operator -(const Point &b)const{
        return Point(x - b.x, y - b.y);
    }

    __device__ int operator <(const Point &b)const{
        if (x == b.x){
            return y >= b.y;
        }
        return x >= b.x;
    }
};

__device__ inline float cross(const Point &a, const Point &b){
    return a.x * b.y - a.y * b.x;
}

__device__ inline float cross(const Point &p1, const Point &p2, const Point &p0){
    return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2, const Point &q1, const Point &q2){
    int ret = min(p1.x,p2.x) <= max(q1.x,q2.x)  &&
              min(q1.x,q2.x) <= max(p1.x,p2.x) &&
              min(p1.y,p2.y) <= max(q1.y,q2.y) &&
              min(q1.y,q2.y) <= max(p1.y,p2.y);
    return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p){
    //params: (7) [x, y, z, dx, dy, dz, heading]
    const float MARGIN = 1e-2;

    float center_x = box[0], center_y = box[1];
    float angle_cos = cos(-box[6]), angle_sin = sin(-box[6]);  // rotate the point in the opposite direction of box
    float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * (-angle_sin);
    float rot_y = (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos;

    return (fabs(rot_x) < box[3] / 2 + MARGIN && fabs(rot_y) < box[4] / 2 + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0, const Point &q1, const Point &q0, Point &ans){
    // fast exclusion
    if (check_rect_cross(p0, p1, q0, q1) == 0) return 0;

    // check cross standing
    float s1 = cross(q0, p1, p0);
    float s2 = cross(p1, q1, p0);
    float s3 = cross(p0, q1, q0);
    float s4 = cross(q1, p1, q0);

    if (!(s1 * s2 > 0 && s3 * s4 > 0)) return 0;

    // calculate intersection of two lines
    float s5 = cross(q1, p1, p0);
    if(fabs(s5 - s1) > EPS){
        ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
        ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);

    }
    else{
        float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
        float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
        float D = a0 * b1 - a1 * b0;

        ans.x = (b0 * c1 - b1 * c0) / D;
        ans.y = (a1 * c0 - a0 * c1) / D;
    }

    return 1;
}

__device__ inline void rotate_around_center(const Point &center, const float angle_cos, const float angle_sin, Point &p){
    float new_x = (p.x - center.x) * angle_cos + (p.y - center.y) * (-angle_sin) + center.x;
    float new_y = (p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
    p.set(new_x, new_y);
}

__device__ inline int point_cmp(const Point &a, const Point &b, const Point &center){
    return atan2(a.y - center.y, a.x - center.x) > atan2(b.y - center.y, b.x - center.x);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b){
    // params box_a: [x, y, z, dx, dy, dz, heading]
    // params box_b: [x, y, z, dx, dy, dz, heading]

    float a_angle = box_a[6], b_angle = box_b[6];
    float a_dx_half = box_a[3] / 2, b_dx_half = box_b[3] / 2, a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2;
    float a_x1 = box_a[0] - a_dx_half, a_y1 = box_a[1] - a_dy_half;
    float a_x2 = box_a[0] + a_dx_half, a_y2 = box_a[1] + a_dy_half;
    float b_x1 = box_b[0] - b_dx_half, b_y1 = box_b[1] - b_dy_half;
    float b_x2 = box_b[0] + b_dx_half, b_y2 = box_b[1] + b_dy_half;

    Point center_a(box_a[0], box_a[1]);
    Point center_b(box_b[0], box_b[1]);

#ifdef DEBUG
    printf("a: (%.3f, %.3f, %.3f, %.3f, %.3f), b: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", a_x1, a_y1, a_x2, a_y2, a_angle,
           b_x1, b_y1, b_x2, b_y2, b_angle);
    printf("center a: (%.3f, %.3f), b: (%.3f, %.3f)\n", center_a.x, center_a.y, center_b.x, center_b.y);
#endif

    Point box_a_corners[5];
    box_a_corners[0].set(a_x1, a_y1);
    box_a_corners[1].set(a_x2, a_y1);
    box_a_corners[2].set(a_x2, a_y2);
    box_a_corners[3].set(a_x1, a_y2);

    Point box_b_corners[5];
    box_b_corners[0].set(b_x1, b_y1);
    box_b_corners[1].set(b_x2, b_y1);
    box_b_corners[2].set(b_x2, b_y2);
    box_b_corners[3].set(b_x1, b_y2);

    // get oriented corners
    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++){
#ifdef DEBUG
        printf("before corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
#ifdef DEBUG
        printf("corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
    }

    box_a_corners[4] = box_a_corners[0];
    box_b_corners[4] = box_b_corners[0];

    // get intersection of lines
    Point cross_points[16];
    Point poly_center;
    int cnt = 0, flag = 0;

    poly_center.set(0, 0);
    for (int i = 0; i < 4; i++){
        for (int j = 0; j < 4; j++){
            flag = intersection(box_a_corners[i + 1], box_a_corners[i], box_b_corners[j + 1], box_b_corners[j], cross_points[cnt]);
            if (flag){
                poly_center = poly_center + cross_points[cnt];
                cnt++;
#ifdef DEBUG
                printf("Cross points (%.3f, %.3f): a(%.3f, %.3f)->(%.3f, %.3f), b(%.3f, %.3f)->(%.3f, %.3f) \n",
                    cross_points[cnt - 1].x, cross_points[cnt - 1].y,
                    box_a_corners[i].x, box_a_corners[i].y, box_a_corners[i + 1].x, box_a_corners[i + 1].y,
                    box_b_corners[i].x, box_b_corners[i].y, box_b_corners[i + 1].x, box_b_corners[i + 1].y);
#endif
            }
        }
    }

    // check corners
    for (int k = 0; k < 4; k++){
        if (check_in_box2d(box_a, box_b_corners[k])){
            poly_center = poly_center + box_b_corners[k];
            cross_points[cnt] = box_b_corners[k];
            cnt++;
#ifdef DEBUG
                printf("b corners in a: corner_b(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
        }
        if (check_in_box2d(box_b, box_a_corners[k])){
            poly_center = poly_center + box_a_corners[k];
            cross_points[cnt] = box_a_corners[k];
            cnt++;
#ifdef DEBUG
                printf("a corners in b: corner_a(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
        }
    }

    poly_center.x /= cnt;
    poly_center.y /= cnt;

    // sort the points of polygon
    Point temp;
    for (int j = 0; j < cnt - 1; j++){
        for (int i = 0; i < cnt - j - 1; i++){
            if (point_cmp(cross_points[i], cross_points[i + 1], poly_center)){
                temp = cross_points[i];
                cross_points[i] = cross_points[i + 1];
                cross_points[i + 1] = temp;
            }
        }
    }

#ifdef DEBUG
    printf("cnt=%d\n", cnt);
    for (int i = 0; i < cnt; i++){
        printf("All cross point %d: (%.3f, %.3f)\n", i, cross_points[i].x, cross_points[i].y);
    }
#endif

    // get the overlap areas
    float area = 0;
    for (int k = 0; k < cnt - 1; k++){
        area += cross(cross_points[k] - cross_points[0], cross_points[k + 1] - cross_points[0]);
    }

    return fabs(area) / 2.0;
}

__device__ inline float box_union(const float *box_a, const float *box_b){

    float a_angle = box_a[6], b_angle = box_b[6];
    float a_dx_half = box_a[3] / 2, b_dx_half = box_b[3] / 2, a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2;
    float a_x1 = box_a[0] - a_dx_half, a_y1 = box_a[1] - a_dy_half;
    float a_x2 = box_a[0] + a_dx_half, a_y2 = box_a[1] + a_dy_half;
    float b_x1 = box_b[0] - b_dx_half, b_y1 = box_b[1] - b_dy_half;
    float b_x2 = box_b[0] + b_dx_half, b_y2 = box_b[1] + b_dy_half;

    Point center_a(box_a[0], box_a[1]);
    Point center_b(box_b[0], box_b[1]);

    Point box_a_corners[4];
    box_a_corners[0].set(a_x1, a_y1);
    box_a_corners[1].set(a_x2, a_y1);
    box_a_corners[2].set(a_x2, a_y2);
    box_a_corners[3].set(a_x1, a_y2);

    Point box_b_corners[4];
    box_b_corners[0].set(b_x1, b_y1);
    box_b_corners[1].set(b_x2, b_y1);
    box_b_corners[2].set(b_x2, b_y2);
    box_b_corners[3].set(b_x1, b_y2);

    // get oriented corners
    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++){
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
    }

    Point total_corners[8];
    for (int k = 0; k < 4; k++){
        total_corners[k] = box_a_corners[k];
        total_corners[k + 4] = box_b_corners[k];
    }

    Point temp;
    for (int k = 0; k < 8; k++){
        for (int i = 0; i < 8 - k - 1; i++){
            if (total_corners[i] < total_corners[i+1]){
                temp = total_corners[i];
                total_corners[i] = total_corners[i + 1];
                total_corners[i + 1] = temp;
            }
        }
    }
    // for (int k = 0; k < 8; k++){
    //     printf("All point %d: (%.3f, %.3f)\n", k, total_corners[k].x, total_corners[k].y);
    // }

    int pos = 1;
    int hull[9];
    int used[8] = {0};

    for (int k = 1; k < 8; k++){
        while(pos > 1 && cross(total_corners[hull[pos - 1]], total_corners[k], total_corners[hull[pos - 2]]) <= 0){
            used[hull[pos -1]] = 0;
            pos--;
        }
        used[k] = 1;
        hull[pos++] = k;
    }

    int m = pos;
    for (int k = 8 - 2; k >= 0; k--){
        if (!used[k]){
            while(pos > m && cross(total_corners[hull[pos - 1]], total_corners[k], total_corners[hull[pos - 2]]) <= 0){
                used[hull[pos -1]] = 0;
                pos--;
            }
            used[k] = 1;
            hull[pos++] = k;
        }
    }

    pos--;
    Point hull_pts[8];
    for (int k = 0; k < pos; k++){
        hull_pts[k] = total_corners[hull[k]];
        // printf("convex hull toto:%d %d %d: (%.3f, %.3f)\n", k, used[k], hull[k], hull_pts[k].x, hull_pts[k].y);
    }

    float area = 0;
    for (int k = 0; k < pos - 1; k++){
        area += cross(hull_pts[k] - hull_pts[0], hull_pts[k + 1] - hull_pts[0]);
    }

    return fabs(area) / 2.0;
}

__device__ inline float iou_bev(const float *box_a, const float *box_b){
    // params box_a: [x, y, z, dx, dy, dz, heading]
    // params box_b: [x, y, z, dx, dy, dz, heading]
    float sa = box_a[3] * box_a[4];
    float sb = box_b[3] * box_b[4];
    float s_overlap = box_overlap(box_a, box_b);
    return s_overlap / fmaxf(sa + sb - s_overlap, EPS);
}

__global__ void boxes_overlap_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap){
    // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
    // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
    const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    if (a_idx >= num_a || b_idx >= num_b){
        return;
    }
    const float * cur_box_a = boxes_a + a_idx * 7;
    const float * cur_box_b = boxes_b + b_idx * 7;
    float s_overlap = box_overlap(cur_box_a, cur_box_b);
    ans_overlap[a_idx * num_b + b_idx] = s_overlap;
}

__global__ void boxes_union_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_union){
    // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
    // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
    const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    if (a_idx >= num_a || b_idx >= num_b){
        return;
    }

    const float * cur_box_a = boxes_a + a_idx * 7;
    const float * cur_box_b = boxes_b + b_idx * 7;
    float s_union = box_union(cur_box_a, cur_box_b);

    // printf("%d %d area: %.3f\n", a_idx, b_idx, s_union);
    ans_union[a_idx * num_b + b_idx] = s_union;
}

__global__ void nms_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask){
    //params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
    //params: mask (N, N/THREADS_PER_BLOCK_NMS)

    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    // if (row_start > col_start) return;

    const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
    const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

    __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
        block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
        block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
        block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
        block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
        block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
        block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
        const float *cur_box = boxes + cur_box_idx * 7;

        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
          start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (iou_bev(cur_box, block_boxes + i * 7) > nms_overlap_thresh){
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
        mask[cur_box_idx * col_blocks + col_start] = t;
    }
}


void nmsLauncher(const float *boxes, unsigned long long * mask, int boxes_num, float nms_overlap_thresh){
    dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
                DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
    dim3 threads(THREADS_PER_BLOCK_NMS);
    nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}

void boxesoverlapLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap){

    dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    boxes_overlap_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_overlap);
}

void boxesunionLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_union){

    dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    boxes_union_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_union);
}